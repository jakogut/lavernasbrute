#include "hip/hip_runtime.h"
// PArt of Laverna's Brute

#include "MD4_CUDA.cuh"

__global__ void cuda_encrypt(unsigned int* wd, unsigned int* message)
{
	int i;

	unsigned int tidx = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ unsigned int s_data[(threadsPerBlock * 4) + (threadsPerBlock * 16)];

	unsigned int* s_wd = s_data;
	unsigned int* s_message = s_wd + threadsPerBlock * 4;

	for(i = 0; i < 4; i++)
		s_wd[tidx * 4 + i] = wd[idx + i];


	for(i = 0; i < 16; i++)
		s_message[tidx * 16 + i] = message[idx + i];

	/* Here, we create aliases to the four 32-bit words used during the encryption process.
	"*a", "*b", "*c", and "*d" are a little easier to read than s_wd[tidx + X]... 		*/
	unsigned int *a = &s_wd[tidx + 0], *b = &s_wd[tidx + 1], *c = &s_wd[tidx + 2], *d = &s_wd[tidx + 3];

	/* Round 1 */
	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 0]  ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 1]  ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 2]  ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 3]  ;*b = (*b << 19) | (*b >> 13);

	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 0]  ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 1]  ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 2]  ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 3]  ;*b = (*b << 19) | (*b >> 13);

	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 8]  ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 9]  ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 10] ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 11] ;*b = (*b << 19) | (*b >> 13);

	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 12] ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 13] ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 14] ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 15] ;*b = (*b << 19) | (*b >> 13);

	/* Round 2 */
	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 0] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 4] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 8] +SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 12]+SQRT_2; *b = (*b<<13) | (*b>>19);

	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 1] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 5] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 9] +SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 13]+SQRT_2; *b = (*b<<13) | (*b>>19);

	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 2] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 6] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 10]+SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 14]+SQRT_2; *b = (*b<<13) | (*b>>19);

	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 3] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 7] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 11]+SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 15]+SQRT_2; *b = (*b<<13) | (*b>>19);

	/* Round 3 */
	*a += (*d ^ *c ^ *b) + s_message[tidx + 0]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 8]  +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 4]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 12] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	*a += (*d ^ *c ^ *b) + s_message[tidx + 2]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 10] +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 6]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 14] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	*a += (*d ^ *c ^ *b) + s_message[tidx + 1]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 9]  +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 5]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 13] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	*a += (*d ^ *c ^ *b) + s_message[tidx + 3]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 11] +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 7]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 15] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	// All done. Copy from shared memory back to global.
	for(i = 0; i < 4; i++)
		wd[idx + i] = s_wd[tidx + i];
}
