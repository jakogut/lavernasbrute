// Part of Laverna's Brute

#include "MD4_CUDA.cuh"
#include "MD4_CUDA_kernel.cu"

MD4_CUDA::MD4_CUDA()
{
}

MD4_CUDA::~MD4_CUDA()
{
}

void MD4_CUDA::getHashContext(hashContext* ctx, unsigned int n)
{
	wd_h = (unsigned*)malloc(sizeof(unsigned) * 4 * n);
	message_h = (unsigned*)malloc(sizeof(unsigned) * 16 * n);

	hipMalloc(&wd_d, sizeof(unsigned) * 4 * n);
	hipMalloc(&message_d, sizeof(unsigned) * 16 * n);

	initialize(ctx, n);

	for(int i = 0; i < n; i++)
	{
		for(int j = 0; j < 4; j++)
			wd_h[i*4+j] = ctx[i].wd[j];

		for(int j = 0; j < 16; j++)
			message_h[i*4+j] = ctx[i].message.uint32[j];
	}

	hipMemcpy(wd_d, wd_h, sizeof(unsigned) * 4 * n, hipMemcpyHostToDevice);
	hipMemcpy(message_d, message_h, sizeof(unsigned) * 16 * n, hipMemcpyHostToDevice);

	cuda_encrypt<<<threadsPerBlock, blocksPerGrid>>>(wd_d, message_d);

	hipMemcpy(wd_h, wd_d, sizeof(unsigned) * 4 * n, hipMemcpyDeviceToHost);

	for(int i = 0; i < n; i++)
		for(int j = 0; j < 4; j++)
			ctx[i].wd[j] = wd_h[i*4+j];

	hipFree(message_d);
	hipFree(wd_d);

	free(message_h);
	free(wd_h);
}

void MD4_CUDA::initialize(hashContext* ctx, int n)
{
	for(int i = 0; i < n; i++) memcpy(ctx[i].wd, wd_init, sizeof(unsigned) * 4);
}
