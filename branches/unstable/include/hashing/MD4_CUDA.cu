// Part of Laverna's Brute

#include "MD4_CUDA.cuh"
#include "MD4_CUDA_kernel.cu"

MD4_CUDA::MD4_CUDA()
{
}

MD4_CUDA::~MD4_CUDA()
{
}

void MD4_CUDA::getHashContext(hashContext* ctx, unsigned int n)
{
	wd_h = (unsigned*)malloc(sizeof(unsigned) * 4 * n);
	message_h = (unsigned*)malloc(sizeof(unsigned) * 16 * n);

	hipMalloc(&wd_d, sizeof(unsigned) * 4 * n);
	hipMalloc(&message_d, sizeof(unsigned) * 16 * n);

	for(int i = 0; i < threadsPerBlock * blocksPerGrid; i++)
		for(int j = 0; j < 16; j++)
			message_h[i*16+j] = ctx[i].message.uint32[j];

	cuda_init<<<threadsPerBlock, blocksPerGrid>>>(wd_d);
	hipMemcpy(message_d, message_h, sizeof(unsigned) * 16 * n, hipMemcpyHostToDevice);

	cuda_encrypt<<<threadsPerBlock, blocksPerGrid>>>(wd_d, message_d);

	hipMemcpy(wd_h, wd_d, sizeof(unsigned) * 4 * n, hipMemcpyDeviceToHost);

	for(int i = 0; i < threadsPerBlock * blocksPerGrid; i++)
		for(int j = 0; j < 4; j++)
			ctx[i].wd[j] = wd_h[i*4+j];

	hipFree(message_d);
	hipFree(wd_d);

	free(message_h);
	free(wd_h);
}
