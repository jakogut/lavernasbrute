/* Slightly modified for LB, orginal code written by Alain Espinosa <alainesp@gmail.com>. Thanks, Alain! */

#ifndef MD4_H_
#define MD4_H_

#include <cstring>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define itoa16 "0123456789abcdef"

#define SQRT_2 0x5a827999
#define SQRT_3 0x6ed9eba1

#define threadsPerBlock 128
#define blocksPerGrid 1024

__global__ void cuda_encrypt(unsigned int* wd, unsigned int* message)
{
	int i;

	unsigned int tidx = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ unsigned int s_data[(threadsPerBlock * 4) + (threadsPerBlock * 16)];

	unsigned int* s_wd = s_data;
	unsigned int* s_message = s_wd + threadsPerBlock * 4;

	for(i = 0; i < 4; i++)
		s_wd[tidx + i] = wd[idx + i];


	for(i = 0; i < 16; i++)
		s_message[tidx + i] = message[idx + i];

	/* Here, we create aliases to the four 32-bit words used during the encryption process.
	"*a", "*b", "*c", and "*d" are a little easier to read than s_wd[tidx + X]... 		*/
	unsigned int *a = &s_wd[tidx + 0], *b = &s_wd[tidx + 1], *c = &s_wd[tidx + 2], *d = &s_wd[tidx + 3];

	/* Round 1 */
	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 0]  ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 1]  ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 2]  ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 3]  ;*b = (*b << 19) | (*b >> 13);

	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 0]  ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 1]  ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 2]  ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 3]  ;*b = (*b << 19) | (*b >> 13);

	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 8]  ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 9]  ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 10] ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 11] ;*b = (*b << 19) | (*b >> 13);

	*a += (*d ^ (*b & (*c ^ *d)))  +  s_message[tidx + 12] ;*a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ (*a & (*b ^ *c)))  +  s_message[tidx + 13] ;*d = (*d << 7 ) | (*d >> 25);
	*c += (*b ^ (*d & (*a ^ *b)))  +  s_message[tidx + 14] ;*c = (*c << 11) | (*c >> 21);
	*b += (*a ^ (*c & (*d ^ *a)))  +  s_message[tidx + 15] ;*b = (*b << 19) | (*b >> 13);

	/* Round 2 */
	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 0] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 4] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 8] +SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 12]+SQRT_2; *b = (*b<<13) | (*b>>19);

	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 1] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 5] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 9] +SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 13]+SQRT_2; *b = (*b<<13) | (*b>>19);

	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 2] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 6] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 10]+SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 14]+SQRT_2; *b = (*b<<13) | (*b>>19);

	*a += ((*b & (*c | *d)) | (*c & *d)) + s_message[tidx + 3] +SQRT_2; *a = (*a<<3 ) | (*a>>29);
	*d += ((*a & (*b | *c)) | (*b & *c)) + s_message[tidx + 7] +SQRT_2; *d = (*d<<5 ) | (*d>>27);
	*c += ((*d & (*a | *b)) | (*a & *b)) + s_message[tidx + 11]+SQRT_2; *c = (*c<<9 ) | (*c>>23);
	*b += ((*c & (*d | *a)) | (*d & *a)) + s_message[tidx + 15]+SQRT_2; *b = (*b<<13) | (*b>>19);

	/* Round 3 */
	*a += (*d ^ *c ^ *b) + s_message[tidx + 0]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 8]  +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 4]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 12] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	*a += (*d ^ *c ^ *b) + s_message[tidx + 2]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 10] +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 6]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 14] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	*a += (*d ^ *c ^ *b) + s_message[tidx + 1]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 9]  +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 5]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 13] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	*a += (*d ^ *c ^ *b) + s_message[tidx + 3]  +  SQRT_3; *a = (*a << 3 ) | (*a >> 29);
	*d += (*c ^ *b ^ *a) + s_message[tidx + 11] +  SQRT_3; *d = (*d << 9 ) | (*d >> 23);
	*c += (*b ^ *a ^ *d) + s_message[tidx + 7]  +  SQRT_3; *c = (*c << 11) | (*c >> 21);
	*b += (*a ^ *d ^ *c) + s_message[tidx + 15] +  SQRT_3; *b = (*b << 15) | (*b >> 17);

	// All done. Copy from shared memory back to global.
	for(i = 0; i < 4; i++)
		wd[idx + i] = s_wd[tidx + i];
}

typedef struct ctx
{
		unsigned int wd[4];

		union
		{
			unsigned int uint32[16];
			unsigned char uint8[64];
		} message;

} hashContext;

class MD4
{
public:

	MD4()
	{
		wd_init[0] = 0x67452301;
		wd_init[1] = 0xefcdab89;
		wd_init[2] = 0x98badcfe;
		wd_init[3] = 0x10325476;
	}

	~MD4()
	{
	}

	typedef void (MD4::*scalarHashPtr)(hashContext*);

	inline void getHashContext(hashContext* ctx, unsigned int n)
	{
		wd_h = (unsigned*)malloc(sizeof(unsigned) * 4 * n);
		message_h = (unsigned*)malloc(sizeof(unsigned) * 16 * n);

		hipMalloc(&wd_d, sizeof(unsigned) * 4 * n);
		hipMalloc(&message_d, sizeof(unsigned) * 16 * n);

		initialize(ctx, n);

		for(int i = 0; i < n; i++)
		{
			for(int j = 0; j < 4; j++)
				wd_h[i*4+j] = ctx[i].wd[j];

			for(int j = 0; j < 16; j++)
				message_h[i*4+j] = ctx[i].message.uint32[j];
		}

		hipMemcpy(wd_d, wd_h, sizeof(unsigned) * 4 * n, hipMemcpyHostToDevice);
		hipMemcpy(message_d, message_d, sizeof(unsigned) * 16 * n, hipMemcpyHostToDevice);

		cuda_encrypt<<<threadsPerBlock, blocksPerGrid>>>(wd_d, message_d);

		hipMemcpy(wd_h, wd_d, sizeof(unsigned) * 4 * n, hipMemcpyDeviceToHost);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < 4; j++)
				ctx[i].wd[j] = wd_h[i*4+j];
	}

protected:

	inline virtual void initialize(hashContext* ctx, int n)
	{
		for(int i = 0; i < n; i++) memcpy(ctx->wd, wd_init, 4 * sizeof(unsigned));
	}

	unsigned int wd_init[4];

	unsigned int* wd_h;
	unsigned int* message_h;

	unsigned int* wd_d;
	unsigned int* message_d;
};

#endif

