#include "hip/hip_runtime.h"
// PArt of Laverna's Brute

#ifndef MD4_CUDA_KERNEL_CUH_
#define MD4_CUDA_KERNEL_CUH_

#include "MD4_CUDA.cuh"

#define F(X, Y, Z) ((X & Y) | (~X & Z))
#define G(X, Y, Z) ((X & Y) | (X & Z) | (Y & Z))
#define H(X, Y, Z) (X ^ Y ^ Z)

#define ROTL(NUM, PLACES, SIZE) ((NUM << PLACES) | (NUM >> (SIZE - PLACES)))
#define ROTR(NUM, PLACES, SIZE) ((NUM >> PLACES) | (NUM << (SIZE - PLACES)))

#define wd(N) (wd[idx * 4 + N])
#define message(N) (message[idx * 16 + N])

__global__ void cuda_init(unsigned int* wd)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	wd[idx * 4 + 0] = 0x67452301;
	wd[idx * 4 + 1] = 0xefcdab89;
	wd[idx * 4 + 2] = 0x98badcfe;
	wd[idx * 4 + 3] = 0x10325476;
}

__global__ void cuda_encrypt(unsigned int* wd, unsigned int* message)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	wd(0) += F(wd(1), wd(2), wd(3))  +  message(0),  wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += F(wd(0), wd(1), wd(2))  +  message(1),  wd(3) = ROTL(wd(3), 7, 32);
	wd(2) += F(wd(3), wd(0), wd(1))  +  message(2),  wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += F(wd(2), wd(3), wd(0))  +  message(3),  wd(1) = ROTL(wd(1), 19, 32);

	wd(0) += F(wd(1), wd(2), wd(3))  +  message(4),  wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += F(wd(0), wd(1), wd(2))  +  message(5),  wd(3) = ROTL(wd(3), 7, 32);
	wd(2) += F(wd(3), wd(0), wd(1))  +  message(6),  wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += F(wd(2), wd(3), wd(0))  +  message(7),  wd(1) = ROTL(wd(1), 19, 32);

	wd(0) += F(wd(1), wd(2), wd(3))  +  message(8),  wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += F(wd(0), wd(1), wd(2))  +  message(9),  wd(3) = ROTL(wd(3), 7, 32);
	wd(2) += F(wd(3), wd(0), wd(1))  +  message(10), wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += F(wd(2), wd(3), wd(0))  +  message(11), wd(1) = ROTL(wd(1), 19, 32);

	wd(0) += F(wd(1), wd(2), wd(3))  +  message(12), wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += F(wd(0), wd(1), wd(2))  +  message(13), wd(3) = ROTL(wd(3), 7, 32);
	wd(2) += F(wd(3), wd(0), wd(1))  +  message(14), wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += F(wd(2), wd(3), wd(0))  +  message(15), wd(1) = ROTL(wd(1), 19, 32);

	// Round 2 // ---

	wd(0) += G(wd(1), wd(2), wd(3)) + message(0) +SQRT_2, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += G(wd(0), wd(1), wd(2)) + message(4) +SQRT_2, wd(3) = ROTL(wd(3), 5, 32);
	wd(2) += G(wd(3), wd(0), wd(1)) + message(8) +SQRT_2, wd(2) = ROTL(wd(2), 9, 32);
	wd(1) += G(wd(2), wd(3), wd(0)) + message(12)+SQRT_2, wd(1) = ROTL(wd(1), 13, 32);

	wd(0) += G(wd(1), wd(2), wd(3)) + message(1) +SQRT_2, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += G(wd(0), wd(1), wd(2)) + message(5) +SQRT_2, wd(3) = ROTL(wd(3), 5, 32);
	wd(2) += G(wd(3), wd(0), wd(1)) + message(9) +SQRT_2, wd(2) = ROTL(wd(2), 9, 32);
	wd(1) += G(wd(2), wd(3), wd(0)) + message(13)+SQRT_2, wd(1) = ROTL(wd(1), 13, 32);

	wd(0) += G(wd(1), wd(2), wd(3)) + message(2) +SQRT_2, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += G(wd(0), wd(1), wd(2)) + message(6) +SQRT_2, wd(3) = ROTL(wd(3), 5, 32);
	wd(2) += G(wd(3), wd(0), wd(1)) + message(10)+SQRT_2, wd(2) = ROTL(wd(2), 9, 32);
	wd(1) += G(wd(2), wd(3), wd(0)) + message(14)+SQRT_2, wd(1) = ROTL(wd(1), 13, 32);

	wd(0) += G(wd(1), wd(2), wd(3)) + message(3) +SQRT_2, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += G(wd(0), wd(1), wd(2)) + message(7) +SQRT_2, wd(3) = ROTL(wd(3), 5, 32);
	wd(2) += G(wd(3), wd(0), wd(1)) + message(11)+SQRT_2, wd(2) = ROTL(wd(2), 9, 32);
	wd(1) += G(wd(2), wd(3), wd(0)) + message(15)+SQRT_2, wd(1) = ROTL(wd(1), 13, 32);

	// Round 3 // --

	wd(0) += H(wd(3), wd(2), wd(1)) + message(0)  +  SQRT_3, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += H(wd(2), wd(1), wd(0)) + message(8)  +  SQRT_3, wd(3) = ROTL(wd(3), 9, 32);
	wd(2) += H(wd(1), wd(0), wd(3)) + message(4)  +  SQRT_3, wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += H(wd(0), wd(3), wd(2)) + message(12) +  SQRT_3, wd(1) = ROTL(wd(1), 15, 32);

	wd(0) += H(wd(3), wd(2), wd(1)) + message(2)  +  SQRT_3, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += H(wd(2), wd(1), wd(0)) + message(10) +  SQRT_3, wd(3) = ROTL(wd(3), 9, 32);
	wd(2) += H(wd(1), wd(0), wd(3)) + message(6)  +  SQRT_3, wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += H(wd(0), wd(3), wd(2)) + message(14) +  SQRT_3, wd(1) = ROTL(wd(1), 15, 32);

	wd(0) += H(wd(3), wd(2), wd(1)) + message(1)  +  SQRT_3, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += H(wd(2), wd(1), wd(0)) + message(9)  +  SQRT_3, wd(3) = ROTL(wd(3), 9, 32);
	wd(2) += H(wd(1), wd(0), wd(3)) + message(5)  +  SQRT_3, wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += H(wd(0), wd(3), wd(2)) + message(13) +  SQRT_3, wd(1) = ROTL(wd(1), 15, 32);

	wd(0) += H(wd(3), wd(2), wd(1)) + message(3)  +  SQRT_3, wd(0) = ROTL(wd(0), 3, 32);
	wd(3) += H(wd(2), wd(1), wd(0)) + message(11) +  SQRT_3, wd(3) = ROTL(wd(3), 9, 32);
	wd(2) += H(wd(1), wd(0), wd(3)) + message(7)  +  SQRT_3, wd(2) = ROTL(wd(2), 11, 32);
	wd(1) += H(wd(0), wd(3), wd(2)) + message(15) +  SQRT_3, wd(1) = ROTL(wd(1), 15, 32);
}

#endif
